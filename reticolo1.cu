#include "hip/hip_runtime.h"
#include <inttypes.h>
#include "header.h"


//Thread block size
#define BLOCK_SIZE 16 //number of threads for each block
#define BIN_SIZE 2 //size of a hisogram bin
#define TH 50 //threshold for triggering the histogram

__global__ void GridKernel(const DataFrame, DataFrame, unsigned int*); //aggiungere i parametri



//------------------------------------------------------------
//Histogram algorithm HOST CODE
unsigned int CircleFit(const DataFrame data, DataFrame circles)
{
  //load data to device memory
  DataFrame d_data;
  d_data.w = data.w;
  d_data.h = data.h;
  size_t size = data.w * data.h * sizeof(float);
  hipError_t err = hipMalloc(&d_data.e, size);
  if(err)
    printf("CUDA malloc data DataFrame: %s\n",hipGetErrorString(err));
  hipMemcpy(d_data.e, data.e, size, hipMemcpyHostToDevice);

  //allocate circles
  DataFrame d_circles;
  d_circles.w = circles.w;
  d_circles.h = circles.h;
  size = circles.w * circles.h * sizeof(float);
  err = hipMalloc(&d_circles.e, size);
  if(err)
    printf("CUDA malloc circles DataFrame: %s\n",hipGetErrorString(err));

  //allocate variable for counting circles found  
  unsigned int *d_counter;
  unsigned int  h_counter = 0;

  hipMallocManaged(&d_counter, sizeof(unsigned int));
    if(err)
    printf("CUDA malloc counter variable: %s\n",hipGetErrorString(err));

  // Hai assegnato al puntatore l'indirizzo di memoria da utilizzare, ma e' memoria della GPU, non puoi accedere da qui
  // ma il valore nella cella puntata non e' inizializzato
  hipMemset(d_counter, 0, sizeof(unsigned int));
  hipMemcpy(&h_counter, d_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);

  float time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //start time
  hipEventRecord(start);

   // Define the geometry
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(GRID_SIZE/ dimBlock.x, GRID_SIZE/ dimBlock.y);
      //warning GRID_SIZE must be multiple of 16

  // Invoke kernel
  GridKernel<<<dimGrid, dimBlock>>>(d_data, d_circles, d_counter);
  err = hipDeviceSynchronize();

  //stop time
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  if(err)
    printf("Run kernel: %s\n", hipGetErrorString(err));

  //printf("Time: %3.5f ms\n",time);

  // Read C from device memory
  size = circles.w * circles.h * sizeof(float);
  err = hipMemcpy(circles.e, d_circles.e, size, hipMemcpyDeviceToHost);
  if(err)
    printf("Copy circles off of device: %s\n",hipGetErrorString(err));

  hipMemcpy(&h_counter, d_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);
  //printf("cerchi trovati : %u\n", h_counter);


  // Free device memory
  hipFree(d_data.e);
  hipFree(d_circles.e);
  hipFree(d_counter);

  return h_counter;
} //END HOST FUNCTION



//---------------------------------------------------------
// thread aware log function
__device__ void log_msg(const char * message)
{
  printf("%d.%d.%d.%d-%s", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, message);
}


//Device function to calculate distance
__device__ float Distance2(float x, float y, float xd, float yd)
{
  return (x-xd)*(x-xd)+(y-yd)*(y-yd);
}


//---------------------------------------------------------
//Device function to Get Element from data
__device__ float GetElement(const DataFrame D, int row, int col)
{
  if ( (row < D.h ) && (col < D.w ) )
    return D.e[row * D.w + col];
  else
    return 0.0;
}


//---------------------------------------------------------------
//Device function to Set Element of data
__device__ void SetElement(DataFrame D, int row, int col, float value) 
{
  if ( (row < D.h ) && (col < D.w ) )
   D.e[row * D.w + col] = value;
}


//---------------------------------------------------------
//Device function to fill histo
// __device__ void HistoFill(int x, int y, DataFrame data,int *h, DataFrame circles, unsigned int *counter)
__device__ void HstFill(int x, int y, DataFrame data,DataFrame circles, unsigned int *counter)
{
  const size_t HST_SIZE = GRID_SIZE*2/BIN_SIZE;
  int 	       hst[HST_SIZE] = {0}; // va inizializzato
  int 	       i, idx;

  // atomicAdd(counter,1);

  for ( i=0; i<data.h; i++)
  {
    // float xd, yd, d;
    float xd, yd;
    int d;

    xd = GetElement(data, i, 0);
    yd = GetElement(data, i, 1);
    d  = __float2int_rn( sqrt( Distance2( __int2float_rn(x), __int2float_rn(y),xd,yd) ))/ BIN_SIZE ;

    if (d < HST_SIZE) hst[d] +=1; // questo if serve per evitare errori di accesso alla memoria nel caso di valori "strani"
  }


  // dopo aver riempito l'array hst, scandisce tutti gli elementi e aggiunge alla lista dei cerchi il cerchio trovato
  for ( i=0; i < HST_SIZE; i++)
  {
    if(hst[i]>TH)
    {
      idx = atomicAdd(counter,1); // garantisce che l'incremento avvenga un thread alla volta, quindi idx puo' essere usato solo dal thread in esecuzione

      SetElement(circles, idx,0, __int2float_rn(x));
      SetElement(circles, idx,1, __int2float_rn(y));
      SetElement(circles, idx,2,  __int2float_rn(i*BIN_SIZE) );
    }
  }
}




//---------------------------------------------------------
//Histogram algorithm kernel
__global__ void GridKernel(DataFrame data, DataFrame circles,unsigned int *counter)
{
  //x and y indexes
  int x = blockIdx.x * blockDim.x+ threadIdx.x - (GRID_SIZE / 2);
  int y = blockIdx.y * blockDim.y +threadIdx.y - (GRID_SIZE / 2);

  HstFill(x,y, data, circles, counter);
}


//------------------------------------------------------------
//dump dataframe function
void dump(DataFrame m)
{
  for (int i = 0; i< m.h; i++)     // Loop sulle righe
  {
    for (int j = 0; j< m.w; j++)    // Loop sulle colonne
      printf("%3.1f\t", m.e[i*m.w + j]);
    printf("\n");  // A capo di fine riga
  }
  printf("\n");
}



//-------------------------------------------------------------
int main(int argc, char** argv)
{
  std::vector<float> buf = ReadData("file.dat");

  DataFrame data, circles;

  data.h = buf.size()>>1;
  data.w = 2;
  data.e = (float*) malloc(data.w * data.h * sizeof(float));

  circles.h = 10;
  circles.w = 3;
  circles.e = (float*) malloc(circles.w * circles.h * sizeof(float));

  data.e = &buf[0]; //copy imported floats from buf to data .... non stai copiando i dati, se fai cosi' non ti serve la memoria che hai allocato prima
  buf.clear();

  unsigned int h_counter = CircleFit(data, circles);

  circles.h = h_counter;
  circles.e = (float*) malloc(circles.w * circles.h * sizeof(float));

  h_counter = CircleFit(data, circles); //CircleFit ritorna il numero di cerchi trovati

  buf = media1 (circles); //sposto i dati dei cerchi fittati nel buffer
  if (h_counter == 0 ) buf = {0, 0, 0};
  AppendData (buf, "fit.dat"); //scrivi buf su fit.dat (in append)

//  dump(data);
//  dump(circles);

  return 0;
}
